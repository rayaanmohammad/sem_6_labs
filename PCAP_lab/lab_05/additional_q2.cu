#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void sortRowsKernel(int *matrix, int N, int M) {
    int rowIdx = threadIdx.x;

    if(rowIdx<M){
        int i, j, minIndex;
        int temp;

        for (i = 0; i < N - 1; i++) {
            minIndex = i;
            for (j = i + 1; j < N; j++) {
                if (matrix[rowIdx * N + j] < matrix[rowIdx * N + minIndex]) {
                    minIndex = j;
                }
            }
            temp = matrix[rowIdx * N + minIndex];
            matrix[rowIdx * N + minIndex] = matrix[rowIdx * N + i];
            matrix[rowIdx * N + i] = temp;
        }
    }
}

int main() {
    int N, M;

    printf("Enter the number of rows (M): ");
    scanf("%d", &M);
    printf("Enter the number of columns (N): ");
    scanf("%d", &N);

    size_t size = M * N * sizeof(int);

    int *h_matrix = (int *)malloc(size);

    printf("Enter elements for the matrix: \n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &h_matrix[i * N + j]);
        }
    }

    int *d_matrix;
    hipMalloc((void **)&d_matrix, size);

    hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);

    int numThreads = M;
    sortRowsKernel<<<1, numThreads>>>(d_matrix, N, M);

    hipMemcpy(h_matrix, d_matrix, size, hipMemcpyDeviceToHost);

    printf("Sorted matrix: \n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_matrix[i * N + j]);
        }
        printf("\n");
    }

    free(h_matrix);
    hipFree(d_matrix);

    return 0;
}
