#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void convulution_one_dimension(float *N, float *M, float *P, int width, int mask_width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    float pvalue = 0;
    int i = idx - (mask_width/2);
    for(int j = 0 ;j<mask_width;j++){
        if(i+j>=0 && i+j<width){
            pvalue += N[i+j] * M[j];
        }
    }
    P[idx] = pvalue;
}

int main() {
    int width, mask_width;
    
    printf("Enter the length of the width of array N: ");
    scanf("%d",&width);

    size_t size_N = width * sizeof(float);
    float *N = (float *)malloc(size_N);
    float *P = (float *)malloc(size_N);

    printf("Enter elements for array N: ");
    for (int i = 0; i < width; i++) {
        scanf("%f", &N[i]);
    }

    printf("Enter the length of the width of mask array M: ");
    scanf("%d",&mask_width);

    size_t size_M = mask_width * sizeof(float);
    float *M = (float *)malloc(size_M);

    printf("Enter elements for mask M: ");
    for (int i = 0; i < mask_width; i++) {
        scanf("%f", &M[i]);
    }

    float *d_N, *d_M, *d_P;
    hipMalloc((void **)&d_N, size_N);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_P, size_N);

    hipMemcpy(d_N, N, size_N, hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, size_M, hipMemcpyHostToDevice);

    convulution_one_dimension<<<1, width>>>(d_N, d_M, d_P, width, mask_width);

    hipMemcpy(P, d_P, size_N, hipMemcpyDeviceToHost);

    printf("Result of convolution operation: ");
    for (int i = 0; i < width; i++) {
        printf("%f ",P[i]);
    }
    printf("\n");
    
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    free(N);
    free(M);
    free(P);
    return 0;
}
