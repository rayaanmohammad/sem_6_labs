#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void convolution_1d(int *N, int *M, int *P, int width, int mask_width) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < width) {
        int half_mask = mask_width / 2;
        int sum = 0;
        int start_index = idx - half_mask;

        for (int j = 0; j < mask_width; j++) {
            int input_index = start_index + j;
            if (input_index >= 0 && input_index < width) {
                sum += N[input_index] * M[j];
            }
        }
        
        P[idx] = sum;  
    }
}

int main() {
    int width, mask_width;
    printf("Enter the width of input array: ");
    scanf("%d", &width);
    printf("Enter the width of the mask array (odd number): ");
    scanf("%d", &mask_width);

    int *N = (int *)malloc(width * sizeof(int));
    int *M = (int *)malloc(mask_width * sizeof(int));
    int *P = (int *)malloc(width * sizeof(int));

    printf("Enter the input array (size %d):\n", width);
    for (int i = 0; i < width; i++) {
        scanf("%d", &N[i]);
    }

    printf("Enter the mask array (size %d):\n", mask_width);
    for (int i = 0; i < mask_width; i++) {
        scanf("%d", &M[i]);
    }

    int *d_N, *d_M, *d_P;
    hipMalloc((void**)&d_N, width * sizeof(int));
    hipMalloc((void**)&d_M, mask_width * sizeof(int));
    hipMalloc((void**)&d_P, width * sizeof(int));

    hipMemcpy(d_N, N, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (width + threadsPerBlock - 1) / threadsPerBlock;

    convolution_1d<<<blocks, threadsPerBlock>>>(d_N, d_M, d_P, width, mask_width);
    
    hipDeviceSynchronize();

    hipMemcpy(P, d_P, width * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nResulting Convoluted Array:\n");
    for (int i = 0; i < width; i++) {
        printf("%d ", P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
    free(N);
    free(M);
    free(P);

    return 0;
}