#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void oddEvenSort(int *a, int n, int phase) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = idx * 2 + phase;

    if (i + 1 < n) {
        if (a[i] > a[i + 1]) {
            int temp = a[i];
            a[i] = a[i + 1];
            a[i + 1] = temp;
        }
    }
}

int main() {
    int n;
    printf("Enter number of elements: ");
    scanf("%d", &n);

    int *a = (int *)malloc(n * sizeof(int));
    int *result = (int *)malloc(n * sizeof(int));

    printf("Enter array elements:\n");
    for (int i = 0; i < n; i++) {
        scanf("%d", &a[i]);
    }

    int *d_a;
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numThreads = (n + 1) / 2;
    int blocks = (numThreads + threadsPerBlock - 1) / threadsPerBlock;

    for (int phase = 0; phase < n; phase++) {
        oddEvenSort<<<blocks, threadsPerBlock>>>(d_a, n, phase % 2);
        hipDeviceSynchronize();
    }

    hipMemcpy(result, d_a, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSorted array:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", result[i]);
    }
    printf("\n");

    hipFree(d_a);
    free(a);
    free(result);

    return 0;
}
