#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void multiplyRowWise(int* A, int* B, int* C, int N, int M, int P) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < N) {
        for (int j = 0; j < P; j++) {
            int sum = 0;
            for (int k = 0; k < M; k++) {
                sum += A[row * M + k] * B[k * P + j];
            }
            C[row * P + j] = sum;
        }
    }
}

__global__ void multiplyColumnWise(int* A, int* B, int* C, int N, int M, int P) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < P) {
        for (int i = 0; i < N; i++) {
            int sum = 0;
            for (int k = 0; k < M; k++) {
                sum += A[i * M + k] * B[k * P + col];
            }
            C[i * P + col] = sum;
        }
    }
}

__global__ void multiplyElementWise(int* A, int* B, int* C, int N, int M, int P) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < N && col < P) {
        int sum = 0;
        for (int k = 0; k < M; k++) {
            sum += A[row * M + k] * B[k * P + col];
        }
        C[row * P + col] = sum;
    }
}

int main() {
    int N, M, P;
    printf("Enter dimensions (N M P) for matrices A (NxM) and B (MxP): ");
    scanf("%d %d %d", &N, &M, &P);

    int sizeA = N * M * sizeof(int);
    int sizeB = M * P * sizeof(int);
    int sizeC = N * P * sizeof(int);

    int *A, *B, *C;
    A = (int*)malloc(sizeA);
    B = (int*)malloc(sizeB);
    C = (int*)malloc(sizeC);

    printf("Enter elements for matrix A (%d x %d):\n", N, M);
    for (int i = 0; i < N * M; i++) scanf("%d", &A[i]);

    printf("Enter elements for matrix B (%d x %d):\n", M, P);
    for (int i = 0; i < M * P; i++) scanf("%d", &B[i]);

    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    printf("\nRow-wise Multiplication:\n");
    multiplyRowWise<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C, N, M, P);
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < P; j++) {
            printf("%d ", C[i * P + j]);
        }
        printf("\n");
    }

    printf("\nColumn-wise Multiplication:\n");
    multiplyColumnWise<<<(P + 255) / 256, 256>>>(d_A, d_B, d_C, N, M, P);
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < P; j++) {
            printf("%d ", C[i * P + j]);
        }
        printf("\n");
    }

    printf("\nElement-wise Multiplication:\n");
    dim3 grid(N, 1);
    dim3 block(P, 1);
    multiplyElementWise<<<grid, block>>>(d_A, d_B, d_C, N, M, P);
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < P; j++) {
            printf("%d ", C[i * P + j]);
        }
        printf("\n");
    }

    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}