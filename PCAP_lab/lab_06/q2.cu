#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void compute_ranks(float *arr, int *ranks, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        int rank = 0;
        for (int j = 0; j < n; j++) {
            if (arr[j] < arr[idx]) {
                rank++;
            }
        }
        ranks[idx] = rank;
    }
}

void parallel_rank_based_selection_sort(float *arr, int n) {
    float *d_arr;
    int *d_ranks;
    size_t size = n * sizeof(float);
    
    hipMalloc((void **)&d_arr, size);
    hipMalloc((void **)&d_ranks, n * sizeof(int));

    hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);

    compute_ranks<<<(n + 255) / 256, 256>>>(d_arr, d_ranks, n);

    int *ranks = (int *)malloc(n * sizeof(int));
    hipMemcpy(ranks, d_ranks, n * sizeof(int), hipMemcpyDeviceToHost);

    float *temp_arr = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        temp_arr[ranks[i]] = arr[i];
    }

    for (int i = 0; i < n; i++) {
        arr[i] = temp_arr[i];
    }

    free(temp_arr);
    hipFree(d_arr);
    hipFree(d_ranks);
}

int main() {
    int n;

    cout << "Enter the number of elements: ";
    cin >> n;

    float *arr = (float *)malloc(n * sizeof(float));

    cout << "Enter the elements of the array: ";
    for (int i = 0; i < n; i++) {
        cin >> arr[i];
    }

    parallel_rank_based_selection_sort(arr, n);

    cout << "Sorted array: ";
    for (int i = 0; i < n; i++) {
        cout << arr[i] << " ";
    }
    cout << endl;

    free(arr);
    return 0;
}
