#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void oddEvenTranspositionSortKernel(int *d_arr, int N, int phase) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N - 1) {
        if (phase % 2 == 0) {
            if (idx % 2 == 0 && d_arr[idx] > d_arr[idx + 1]) {
                int temp = d_arr[idx];
                d_arr[idx] = d_arr[idx + 1];
                d_arr[idx + 1] = temp;
            }
        } else {
            if (idx % 2 != 0 && d_arr[idx] > d_arr[idx + 1]) {
                int temp = d_arr[idx];
                d_arr[idx] = d_arr[idx + 1];
                d_arr[idx + 1] = temp;
            }
        }
    }
}

int main() {
    int N;
    printf("Enter the number of elements in the array: ");
    scanf("%d", &N);

    size_t size = N * sizeof(int);
    int *h_arr = (int *)malloc(size);

    printf("Enter elements for the array: \n");
    for (int i = 0; i < N; i++) {
        scanf("%d", &h_arr[i]);
    }

    int *d_arr;
    hipMalloc((void **)&d_arr, size);

    hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);

    int numThreadsPerBlock = N;
    int numBlocks = 1;

    for (int phase = 0; phase < N; phase++) {
        oddEvenTranspositionSortKernel<<<numBlocks, numThreadsPerBlock>>>(d_arr, N, phase);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);

    printf("Sorted array: \n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    free(h_arr);
    hipFree(d_arr);
    return 0;
}
