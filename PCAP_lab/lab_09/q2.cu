#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void transformMatrix(int *input, int *output, int rows, int cols) {
    int row = blockIdx.x;
    if (row < rows) {
        int power = row + 1; 
        for (int col = 0; col < cols; col++) {
            int base = input[row * cols + col];
            int result = 1;
            for (int p = 0; p < power; p++) { 
                result *= base;
            }
            output[row * cols + col] = result;
        }
    }
}

int main() {
    int rows, cols;
    printf("Enter number of rows and columns: ");
    scanf("%d %d", &rows, &cols);

    int *h_input = (int*)malloc(rows * cols * sizeof(int));
    int *h_output = (int*)malloc(rows * cols * sizeof(int));

    printf("Enter matrix elements:\n");
    for (int i = 0; i < rows * cols; i++) {
        scanf("%d", &h_input[i]);
    }

    int *d_input, *d_output;
    hipMalloc((void**)&d_input, rows * cols * sizeof(int));
    hipMalloc((void**)&d_output, rows * cols * sizeof(int));

    hipMemcpy(d_input, h_input, rows * cols * sizeof(int), hipMemcpyHostToDevice);

    transformMatrix<<<rows, 1>>>(d_input, d_output, rows, cols);

    hipMemcpy(h_output, d_output, rows * cols * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nOriginal Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", h_input[i * cols + j]);
        }
        printf("\n");
    }

    printf("\nTransformed Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", h_output[i * cols + j]);
        }
        printf("\n");
    }

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
