#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void printhello(){
    int idx = threadIdx.x;
    printf("Hello from thread no: %d\n",idx);
    return;
}

int main(){
    int n = 0;
    printf("Enter the number of threads you want:");
    scanf("%d",&n);
    printhello<<<1,n>>>();
    hipDeviceSynchronize();
    return 0;
}