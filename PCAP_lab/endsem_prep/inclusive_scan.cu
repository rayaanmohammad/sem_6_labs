
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void inclusiveScanKernel(int *d_in, int *d_out, int n) {
    extern __shared__ int temp[]; // shared memory

    int tid = threadIdx.x;

    // Load input into shared memory
    if (tid < n) temp[tid] = d_in[tid];
    __syncthreads();

    // Inclusive scan: Hillis-Steele algorithm
    for (int offset = 1; offset < n; offset *= 2) {
        int val = 0;
        if (tid >= offset) {
            val = temp[tid - offset];
        }
        __syncthreads();
        if (tid < n) {
            temp[tid] += val;
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tid < n) d_out[tid] = temp[tid];
}

int main() {
    const int N = 8;
    int h_in[N] = {1, 2, 3, 4, 5, 6, 7, 8};
    int h_out[N];

    int *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(int));
    hipMalloc(&d_out, N * sizeof(int));

    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 block of N threads and N shared memory size
    inclusiveScanKernel<<<1, N, N * sizeof(int)>>>(d_in, d_out, N);
    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Inclusive scan result:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_out[i]);
    }
    printf("\n");

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
