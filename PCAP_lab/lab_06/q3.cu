#include <stdio.h>
#include <hip/hip_runtime.h>
 
__global__ void odd_phase_kernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (idx % 2 == 1 && idx < n - 1) {
        if (data[idx] > data[idx + 1]) {
            int temp = data[idx];
            data[idx] = data[idx + 1];
            data[idx + 1] = temp;
        }
    }
}
 
__global__ void even_phase_kernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (idx % 2 == 0 && idx < n - 1) {
        if (data[idx] > data[idx + 1]) {
            int temp = data[idx];
            data[idx] = data[idx + 1];
            data[idx + 1] = temp;
        }
    }
}
 
void parallelOddEvenSort(int *h_arr, int n) {
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));
 
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);
 
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
 
    for (int phase = 0; phase < n; ++phase) {
        odd_phase_kernel<<<numBlocks, blockSize>>>(d_arr, n);
        hipDeviceSynchronize();
 
        even_phase_kernel<<<numBlocks, blockSize>>>(d_arr, n);
        hipDeviceSynchronize();
    }
 
    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
 
    hipFree(d_arr);
}
 
int main() {
    const int n = 8;
    int h_arr[n] = {64, 25, 12, 22, 11, 90, 45, 33};
 
    printf("Original array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");
 
    parallelOddEvenSort(h_arr, n);
 
    printf("Sorted array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");
 
    return 0;
}