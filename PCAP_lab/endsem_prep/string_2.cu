#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

__device__ int findStart(int n, int idx) {
    int answer = n * (n + 1) / 2;
    int after = n - idx;
    answer = answer - (after * (after + 1) / 2);
    return answer;
}

__global__ void change_string(char* d_s, char* d_result, int* d_len) {
    int idx = threadIdx.x;
    int len = *d_len;
    int len_2 = len - idx;
    int start_index = findStart(len, idx);

    if (idx < len) {
        for (int i = 0; i < len_2; i++) {
            d_result[start_index + i] = d_s[i];
        }
    }
}

int main() {
    char s[100];
    int len, result_len;

    printf("Enter the string: ");
    scanf("%s", s);

    len = strlen(s);
    result_len = len * (len + 1) / 2;

    char* result = (char*)malloc(result_len * sizeof(char));

    char *d_s, *d_result;
    int *d_len;

    hipMalloc((void**)&d_s, len * sizeof(char));
    hipMalloc((void**)&d_result, result_len * sizeof(char));
    hipMalloc((void**)&d_len, sizeof(int));

    hipMemcpy(d_s, s, len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);

    change_string<<<1, len>>>(d_s, d_result, d_len);
    hipDeviceSynchronize();

    hipMemcpy(result, d_result, result_len * sizeof(char), hipMemcpyDeviceToHost);

    for (int i = 0; i < result_len; i++) {
        printf("%c", result[i]);
    }
    printf("\n");

    hipFree(d_s);
    hipFree(d_result);
    hipFree(d_len);
    free(result);

    return 0;
}
