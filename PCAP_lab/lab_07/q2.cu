#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void copyAndCount(char* input, char* output, int inputLength) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < inputLength) {
        int copyLength = inputLength - idx;
        int outputIdx = (inputLength - (copyLength+1) + 1) * ((copyLength+1) + inputLength) / 2;
        for (int i = 0; i < copyLength; ++i) {
            output[outputIdx + i] = input[i]; 
        }
    }
}

int main() {
    char word[100];
    printf("Enter the word: ");
    scanf("%s", word);
    int inputLength = strlen(word);

    int outputSize = (inputLength * (inputLength + 1)) / 2; 

    char* d_input;
    char* d_output;
    char* h_output = (char*)malloc(outputSize * sizeof(char));

    hipMalloc((void**)&d_input, inputLength * sizeof(char));
    hipMalloc((void**)&d_output, outputSize * sizeof(char)); 
    
    hipMemcpy(d_input, word, inputLength * sizeof(char), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (inputLength + blockSize - 1) / blockSize;
    copyAndCount<<<numBlocks, blockSize>>>(d_input, d_output, inputLength);

    hipMemcpy(h_output, d_output, outputSize * sizeof(char), hipMemcpyDeviceToHost);

    printf("Output String: ");
    for (int i = 0; i < outputSize; i++) {
        printf("%c", h_output[i]);
    }
    printf("\n");

    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
