#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addRowWise(int* A, int* B, int* C, int n, int m) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < n) {
        for (int j = 0; j < m; j++) {
            C[row * m + j] = A[row * m + j] + B[row * m + j];
        }
    }
}

__global__ void addColumnWise(int* A, int* B, int* C, int n, int m) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < m) {
        for (int i = 0; i < n; i++) {
            C[i * m + col] = A[i * m + col] + B[i * m + col];
        }
    }
}

__global__ void addElementWise(int* A, int* B, int* C, int n, int m) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < n && col < m) {
        C[row * m + col] = A[row * m + col] + B[row * m + col];
    }
}

int main() {
    int n, m;
    printf("Enter number of rows (N) and columns (M): ");
    scanf("%d %d", &n, &m);

    int size = n * m * sizeof(int);
    int *A, *B, *C;
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    printf("Enter elements for matrix A (%d x %d):\n", n, m);
    for (int i = 0; i < n * m; i++) scanf("%d", &A[i]);

    printf("Enter elements for matrix B (%d x %d):\n", n, m);
    for (int i = 0; i < n * m; i++) scanf("%d", &B[i]);

    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    printf("\nRow-wise Addition:\n");
    addRowWise<<<(n + 255) / 256, 256>>>(d_A, d_B, d_C, n, m);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", C[i * m + j]);
        }
        printf("\n");
    }

    printf("\nColumn-wise Addition:\n");
    addColumnWise<<<(m + 255) / 256, 256>>>(d_A, d_B, d_C, n, m);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", C[i * m + j]);
        }
        printf("\n");
    }

    printf("\nElement-wise Addition:\n");
    dim3 grid(n, 1);
    dim3 block(m, 1);
    addElementWise<<<grid, block>>>(d_A, d_B, d_C, n, m);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", C[i * m + j]);
        }
        printf("\n");
    }

    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
