
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#define MAX_LEN 1024

__device__ bool matchAt(const char* sentence, const char* word, int pos, int wordLen, int sentLen) {
    if (pos + wordLen > sentLen) return false;
    for (int i = 0; i < wordLen; ++i) {
        if (sentence[pos + i] != word[i]) return false;
    }
    if ((pos + wordLen == sentLen || sentence[pos + wordLen] == ' ') &&
        (pos == 0 || sentence[pos - 1] == ' ')) {
        return true;
    }
    return false;
}

__global__ void countWordOccurrences(const char* sentence, const char* word, int* count, int wordLen, int sentLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < sentLen && matchAt(sentence, word, idx, wordLen, sentLen)) {
        atomicAdd(count, 1);
    }
}

int main() {
    char host_sentence[MAX_LEN];
    char host_word[100];

    printf("Enter a sentence:\n");
    fgets(host_sentence, MAX_LEN, stdin);
    host_sentence[strcspn(host_sentence, "\n")] = '\0';

    printf("Enter the word to count:\n");
    fgets(host_word, sizeof(host_word), stdin);
    host_word[strcspn(host_word, "\n")] = '\0'; 
    int wordLen = strlen(host_word);
    int sentLen = strlen(host_sentence);

    char *dev_sentence, *dev_word;
    int *dev_count;
    int host_count = 0;

    hipMalloc((void**)&dev_sentence, sentLen * sizeof(char));
    hipMalloc((void**)&dev_word, wordLen * sizeof(char));
    hipMalloc((void**)&dev_count, sizeof(int));

    hipMemcpy(dev_sentence, host_sentence, sentLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_word, host_word, wordLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_count, &host_count, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (sentLen + threadsPerBlock - 1) / threadsPerBlock;

    countWordOccurrences<<<blocks, threadsPerBlock>>>(dev_sentence, dev_word, dev_count, wordLen, sentLen);
    hipDeviceSynchronize();

    hipMemcpy(&host_count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("Word \"%s\" occurred %d times in the sentence.\n", host_word, host_count);

    hipFree(dev_sentence);
    hipFree(dev_word);
    hipFree(dev_count);

    return 0;
}
