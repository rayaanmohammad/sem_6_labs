#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int toBinary(int num, char binary[]) {
    int index = 0;

    if (num == 0) {
        binary[0] = '0';
        binary[1] = '\0';
        return 1;
    }

    while (num > 0) {
        binary[index++] = (num % 2) + '0';
        num /= 2;
    }
    binary[index] = '\0';

    for (int i = 0; i < index / 2; i++) {
        char temp = binary[i];
        binary[i] = binary[index - i - 1];
        binary[index - i - 1] = temp;
    }

    return index;
}

__device__ void onesComplement(char binary[], int length) {
    for (int i = 0; i < length; i++) {
        binary[i] = (binary[i] == '0') ? '1' : '0';
    }
}

__device__ int binaryStringToInt(const char binary[], int length) {
    int result = 0;
    for (int i = 0; i < length; i++) {
        result = result * 10 + (binary[i] - '0');
    }
    return result;
}

__global__ void modifyMatrix(int *mat, int *output, int rows, int cols) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    int idx = row * cols + col;

    if (row > 0 && row < rows - 1 && col > 0 && col < cols - 1) {
        char binary[32];
        int length = toBinary(mat[idx], binary);
        onesComplement(binary, length);
        output[idx] = binaryStringToInt(binary, length);
    } else {
        output[idx] = mat[idx]; 
    }
}

int main() {
    int n, m;
    printf("Enter number of rows and columns: ");
    scanf("%d %d", &n, &m);

    int size = n * m * sizeof(int);
    int *A = (int*)malloc(size);
    int *result = (int*)malloc(size);

    printf("Enter elements for matrix (%d x %d):\n", n, m);
    for (int i = 0; i < n * m; i++) {
        scanf("%d", &A[i]);
    }

    int *d_A, *d_result;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_result, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    dim3 grid(n, 1);
    dim3 block(m, 1);
    modifyMatrix<<<grid, block>>>(d_A, d_result, n, m);

    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    printf("\nModified Matrix (Non-border elements replaced with one's complement stored as integer):\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", result[i * m + j]);
        }
        printf("\n");
    }

    free(A);
    free(result);
    hipFree(d_A);
    hipFree(d_result);

    return 0;
}
