#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void vectorAddKernel(int *A, int *B, int *C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N;
    printf("Enter the length of the vectors (N): ");
    scanf("%d", &N);

    size_t size = N * sizeof(int);

    int *A = (int *)malloc(size);
    int *B = (int *)malloc(size);
    int *C = (int *)malloc(size);

    printf("Enter elements for vector A: ");
    for (int i = 0; i < N; i++) {
        scanf("%d", &A[i]);
    }

    printf("Enter elements for vector B: ");
    for (int i = 0; i < N; i++) {
        scanf("%d", &B[i]);
    }

    int *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result of vector addition: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", C[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);
    return 0;
}
