#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sortRow(int *a, int n, int m){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i=0; i<m-1; i++){
        int min = i;
        for(int j=i; j<m; j++){
            if(a[row*m + j]<a[row*m + min])
                min = j;
        }
        int temp = a[row*m + i];
        a[row*m + i] = a[row*m + min];
        a[row*m + min] = temp;
    }
}

int main(){
    int n, m;
    printf("Enter n: ");
    scanf("%d",&n);
    printf("Enter m: ");
    scanf("%d",&m);
    int size = n*m*sizeof(int);
    int *a = (int*)malloc(size);
    int *result = (int*)malloc(size);

    printf("Enter the matrix:\n");
    for(int i = 0; i < n*m; i++){
        scanf("%d",&a[i]);
    }

    int *d_a, *d_result;
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_result,size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    sortRow<<<1,n>>> (d_a, n, m);

    hipMemcpy(result, d_a, size, hipMemcpyDeviceToHost);

    printf("\nResultant matrix:\n");
    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){
            printf("%d ",result[i*m+j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    free(a);
    free(result);

    return 0;
}