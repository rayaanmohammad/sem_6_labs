#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void countWordOccurrences(const char* sentence, const char* word, int* wordCount, int sentenceLength, int wordLength) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < sentenceLength - wordLength + 1) {
        bool match = true;
        for (int i = 0; i < wordLength; ++i) {
            if (sentence[idx + i] != word[i]) {
                match = false;
                break;
            }
        }
        if (match) {
            atomicAdd(wordCount, 1);
        }
    }
}

int main() {
    char sentence[100];
    char word[100];
    
    printf("Enter the sentence: ");
    fgets(sentence, sizeof(sentence), stdin); 
    sentence[strcspn(sentence, "\n")] = '\0';

    printf("Enter the word: ");
    scanf("%s", word);
    
    int sentenceLength = strlen(sentence);
    int wordLength = strlen(word);
    
    int* d_wordCount;
    int h_wordCount = 0;

    hipMalloc((void**)&d_wordCount, sizeof(int));
    hipMemcpy(d_wordCount, &h_wordCount, sizeof(int), hipMemcpyHostToDevice);

    char* d_sentence;
    char* d_word;
    hipMalloc((void**)&d_sentence, sentenceLength + 1);
    hipMalloc((void**)&d_word, wordLength + 1);

    hipMemcpy(d_sentence, sentence, sentenceLength + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength + 1, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (sentenceLength - wordLength + 1 + blockSize - 1) / blockSize;

    countWordOccurrences<<<numBlocks, blockSize>>>(d_sentence, d_word, d_wordCount, sentenceLength, wordLength);

    hipMemcpy(&h_wordCount, d_wordCount, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word, h_wordCount);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_wordCount);

    return 0;
}
