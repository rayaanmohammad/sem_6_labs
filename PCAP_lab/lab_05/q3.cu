#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void computeSine(float *angles, float *sineValues, int N) {
    int idx = threadIdx.x;
    if (idx < N) {
        sineValues[idx] = sin(angles[idx]);
    }
}

int main() {
    int N;
    printf("Enter the number of angles: ");
    scanf("%d", &N);

    size_t size = N * sizeof(float);

    float *h_angles = (float *)malloc(size);
    float *h_sineValues = (float *)malloc(size);

    printf("Enter the angles (in radians): ");
    for (int i = 0; i < N; i++) {
        scanf("%f", &h_angles[i]);
    }

    float *d_angles, *d_sineValues;
    hipMalloc((void **)&d_angles, size);
    hipMalloc((void **)&d_sineValues, size);

    hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);

    computeSine<<<1, N>>>(d_angles, d_sineValues, N);

    hipMemcpy(h_sineValues, d_sineValues, size, hipMemcpyDeviceToHost);

    printf("Sine values of the angles: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_sineValues[i]);
    }
    printf("\n");

    hipFree(d_angles);      
    hipFree(d_sineValues);  

    free(h_angles);          
    free(h_sineValues);
    return 0;
}
