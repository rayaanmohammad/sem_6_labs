#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void linearAlgebraKernel(float *x, float *y, float alpha, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        y[idx] = alpha * x[idx] + y[idx];
    }
}

int main() {
    int N;
    float alpha;

    printf("Enter the length of the vectors (N): ");
    scanf("%d", &N);
    printf("Enter the scalar alpha: ");
    scanf("%f", &alpha);

    size_t size = N * sizeof(float);

    float *X = (float *)malloc(size);
    float *Y = (float *)malloc(size);

    printf("Enter elements for vector x: ");
    for (int i = 0; i < N; i++) {
        scanf("%f", &X[i]);
    }

    printf("Enter elements for vector y: ");
    for (int i = 0; i < N; i++) {
        scanf("%f", &Y[i]);
    }

    float *d_x, *d_y;
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);

    hipMemcpy(d_x, X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, Y, size, hipMemcpyHostToDevice);

    linearAlgebraKernel<<<1, N>>>(d_x, d_y, alpha, N);

    hipMemcpy(Y, d_y, size, hipMemcpyDeviceToHost);

    printf("Resultant vector y: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", Y[i]);
    }
    printf("\n");

    hipFree(d_x);      
    hipFree(d_y);      

    free(X);            
    free(Y);
    return 0;
}
